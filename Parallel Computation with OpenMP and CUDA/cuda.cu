#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Function to check CUDA errors
void hipError_t(hipError_t err, const char* message) {
    if (err != hipSuccess) {
        fprintf(stderr, "Error: %s (code %s)\n", message, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// This function performs heavy computations
// No Changes to this function are allowed
__device__ double heavy(double data, int loopSize) {
    double sum = 0;       
    for (int i = 0; i < loopSize; i++)
        sum += cos(exp(sin(data * (i % 11))))/ loopSize;

     return sum;             
}

__global__ void kernel_heavy(int* size, int* loopSize, double* arr, double* answer) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < *size) {
        double calc = heavy(arr[i], *loopSize);
        atomicAdd(answer, calc);
    }
}
 

// CUDA code to be parallelized
int main(int argc, char *argv[]) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    int i;
    int size = atoi(argv[1]);
    int loopSize = atoi(argv[2]);
    float time = 0;
    hipEvent_t start_time, end_time;

    hipEventCreate(&start_time);
    hipEventCreate(&end_time);

    double* arr = (double*)malloc(size * sizeof(double));
    
    if (!arr) {
        fprintf(stderr, "Memory allocation failed\n");
        exit(EXIT_FAILURE);
    }

    for (i = 0; i < size; i++)
        arr[i] = (double) rand() / RAND_MAX;
    
    // Allocate the device input size
    int *d_size = NULL;
    err = hipMalloc((void **)&d_size, sizeof(int));

    hipError_t(err, "allocate device size");

    // Allocate the device input loop size
    int *d_loopSize = NULL;
    err = hipMalloc((void **)&d_loopSize, sizeof(int));

    hipError_t(err, "allocate device loop size");

    // Allocate the device input vector arr
    double *d_arr = NULL;
    err = hipMalloc((void **)&d_arr, size * sizeof(double));

    hipError_t(err, "allocate device vector arr");

    // Allocate the device output answer
    double *d_answer = NULL;
    err = hipMalloc((void **)&d_answer, sizeof(double));

    hipError_t(err, "allocate device answer");

    hipMemset(d_answer, 0, sizeof(double));

    hipError_t(err, "set asnwer to 0");

    err = hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice);

    hipError_t(err, "copy size from host to device");

    err = hipMemcpy(d_loopSize, &loopSize, sizeof(int), hipMemcpyHostToDevice);

    hipError_t(err, "copy loop size from host to device");

    err = hipMemcpy(d_arr, arr, size * sizeof(double), hipMemcpyHostToDevice);

    hipError_t(err, "copy vector arr from host to device");

    // Launch CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    double answer = 0;
    hipEventRecord(start_time);
    kernel_heavy<<<blocksPerGrid, threadsPerBlock>>>(d_size, d_loopSize, d_arr, d_answer);
    hipEventRecord(end_time);
    hipEventSynchronize(end_time);
    err = hipGetLastError();

    hipError_t(err, "launch kernel");
    
    // Copy the device result in device memory to the host result in host memory.
    err = hipMemcpy(&answer, d_answer, sizeof(double), hipMemcpyDeviceToHost);

    hipError_t(err, "copy answer from device to host");

    printf("answer = %e\n", answer);
    hipEventElapsedTime(&time, start_time, end_time);   
    printf("Time: %f seconds\n", time/1000.0);

    // Free device global memory
    err = hipFree(d_size);

    hipError_t(err, "allocate device size");
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device size (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_loopSize);

    hipError_t(err, "free device loop size");

    err = hipFree(d_arr);

    hipError_t(err, "free device vector arr");

    err = hipFree(d_answer);

    hipError_t(err, "free device answer");

    // Free host memory
    free(arr);

    // Free events
    hipEventDestroy(start_time);
    hipEventDestroy(end_time);
}